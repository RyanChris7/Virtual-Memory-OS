﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ void init_invert_page_table(VirtualMemory *vm)
{

  for (int i = 0; i < vm->PAGE_ENTRIES; i++)
  {
    vm->invert_page_table[i] = 0x80000000;                 // invalid := MSB is 1
    vm->invert_page_table[i + vm->PAGE_ENTRIES] = i;       // Page number
    vm->invert_page_table[i + (2 * vm->PAGE_ENTRIES)] = 0; // Count variable which stores the access time which is used in the recently_used
  }
}

__device__ void vm_init(VirtualMemory *vm, uchar *buffer, uchar *storage,
                        u32 *invert_page_table, int *pagefault_num_ptr,
                        int PAGESIZE, int INVERT_PAGE_TABLE_SIZE,
                        int PHYSICAL_MEM_SIZE, int STORAGE_SIZE,
                        int PAGE_ENTRIES)
{
  // init variables
  vm->buffer = buffer;
  vm->storage = storage;
  vm->invert_page_table = invert_page_table;
  vm->pagefault_num_ptr = pagefault_num_ptr;

  // init constants
  vm->PAGESIZE = PAGESIZE;
  vm->INVERT_PAGE_TABLE_SIZE = INVERT_PAGE_TABLE_SIZE;
  vm->PHYSICAL_MEM_SIZE = PHYSICAL_MEM_SIZE;
  vm->STORAGE_SIZE = STORAGE_SIZE;
  vm->PAGE_ENTRIES = PAGE_ENTRIES;

  // before first vm_write or vm_read
  init_invert_page_table(vm);
}

//Function Prototypes
__device__ int recently_used(VirtualMemory *vm, bool page_check, int index);
__device__ int page_search(VirtualMemory *vm, u32 page_number);
__device__ int update(VirtualMemory *vm, bool check, int index, u32 page_number);

// ===== Code Begin ===== //

__device__ int recently_used(VirtualMemory *vm, bool page_check, int index){
    if (page_check)
        return index;
    u32 used = vm->invert_page_table[2 * vm->PAGE_ENTRIES];
    int final_index = 0;
    int i = 1;

    while (i < vm->PAGE_ENTRIES){
        if (used < vm->invert_page_table[i + (2 * vm->PAGE_ENTRIES)])
        {
        used = vm->invert_page_table[i + (2 * vm->PAGE_ENTRIES)];
        final_index = i;
        }
        i++;
    }
    return final_index;
}

__device__ int page_search(VirtualMemory *vm, u32 page_number){ // Function to search and update page
    int i = 0;
    int index = -1;
    bool check = false;

    while(i < vm->PAGE_ENTRIES){  // Page search in main memory
        if (vm->invert_page_table[i + vm->PAGE_ENTRIES] == page_number){
            check = true;
            index = i;
            break;
        }
        i++;
    }

    // Update page
    index = update(vm, check, index, page_number);
    return index;
}

__device__ int update(VirtualMemory *vm, bool check, int index, u32 page_number){
    if (check)
    {
        if (vm->invert_page_table[index] != 0x80000000)
        return index;

        *(vm->pagefault_num_ptr) += 1;
        for (int i = 0; i < vm->PAGESIZE; i++)
            vm->buffer[(index * vm->PAGESIZE) + i] = vm->storage[(page_number * vm->PAGESIZE) + i];
        vm->invert_page_table[index] = 0x00000000;
        return index;
    }
    else {
        bool page_check = false;

        *(vm->pagefault_num_ptr) += 1; // Increase the pagefault number
        for (int i = 0; i < vm->PAGE_ENTRIES; i++)
        {
            if (vm->invert_page_table[i] == 0x80000000)
            {
            page_check = true;
            index = i;
            break;
            }
        }
        index = recently_used(vm, page_check, index);
    }

    // Write data from main memory to disk
    for (int i = 0; i < vm->PAGESIZE; i++)
        vm->storage[vm->invert_page_table[index + vm->PAGE_ENTRIES] * vm->PAGESIZE + i] =
            vm->buffer[index * vm->PAGESIZE + i];

    // Load data from disk to main memory
    for (int i = 0; i < vm->PAGESIZE; i++)
        vm->buffer[(index * vm->PAGESIZE) + i] = vm->storage[(page_number * vm->PAGESIZE) + i];
    vm->invert_page_table[index] = 0x00000000;

    // Update the invert page table
    vm->invert_page_table[index + vm->PAGE_ENTRIES] = page_number;
    return index;
}

__device__ uchar vm_read(VirtualMemory *vm, u32 addr){
    /* Complate vm_read function to read single element from data buffer */
    u32 page_number = addr / vm->PAGESIZE;
    u32 page_offset = addr % vm->PAGESIZE;

    int index = page_search(vm, page_number);
    uchar data = vm->buffer[(index * vm->PAGESIZE) + page_offset]; // Get data from main memory

    for (int i = 0; i < vm->PAGE_ENTRIES; i++)
        vm->invert_page_table[i + (2 * vm->PAGE_ENTRIES)] += 1;
    vm->invert_page_table[index + (2 * vm->PAGE_ENTRIES)] = 0; // Set the accessed page's count ot 0

    return data;
}


__device__ void vm_write(VirtualMemory *vm, u32 addr, uchar value){
    /* Complete vm_write function to write value into data buffer */
    u32 page_number = addr / vm->PAGESIZE;
    u32 page_offset = addr % vm->PAGESIZE;

    int index = page_search(vm, page_number);
    vm->buffer[(index * vm->PAGESIZE) + page_offset] = value; // write data to main memory

    for (int i = 0; i < vm->PAGE_ENTRIES; i++)
        vm->invert_page_table[i + (2 * vm->PAGE_ENTRIES)] += 1;
    vm->invert_page_table[index + (2 * vm->PAGE_ENTRIES)] = 0; // Set the accessed page's count ot 0
}


__device__ void vm_snapshot(VirtualMemory *vm, uchar *results, int offset,
                            int input_size){
    /* Complete snapshot function togther with vm_read to load elements from data to result buffer */
    for (int i = 0; i < input_size; i++)
        results[i + offset] = vm_read(vm, i);
}